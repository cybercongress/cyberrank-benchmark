#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "types.h"

const int CUDA_THREAD_BLOCK_SIZE = 256;

/*****************************************************/
/* KERNEL: RUN SINGLE RANK ITERATION                 */
/*****************************************************/
/* All in links used here are compressed in links    */
/*****************************************************/
__global__
void run_rank_iteration(
    CompressedInLink *inLinks,                            /* all compressed in links */
    double *prevRank, double *rank, uint64_t rankSize,    /* array index - cid index */
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount,  /* array index - cid index */
    double defaultRankWithCorrection,                     /* default rank + inner product correction */
    double dampingFactor
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < rankSize; i += stride) {

        if(inLinksCount[i] == 0) {
            continue;
        }

        double ksum = 0;
        for (uint64_t j = inLinksStartIndex[i]; j < inLinksStartIndex[i] + inLinksCount[i]; j++) {
           ksum = prevRank[inLinks[j].fromIndex] * inLinks[j].weight + ksum;
           //ksum = __fmaf_rn(prevRank[inLinks[j].fromIndex], inLinks[j].weight, ksum);
        }
        rank[i] = ksum * dampingFactor + defaultRankWithCorrection;
        //rank[i] = __fmaf_rn(ksum, dampingFactor, defaultRankWithCorrection);
    }
}


/*****************************************************/
/* KERNEL: DOUBLE ABS FUNCTOR                        */
/*****************************************************/
/* Return absolute value for double                  */
/*****************************************************/
struct absolute_value {
  __device__ double operator()(const double &x) const {
    return x < 0.0 ? -x : x;
  }
};


/*****************************************************/
/* HOST: FINDS MAXIMUM RANKS DIFFERENCE              */
/*****************************************************/
/* Finds maximum rank difference for single element  */
/*                                                   */
/*****************************************************/
double find_max_ranks_diff(double *prevRank, double *newRank, uint64_t rankSize) {

    thrust::device_vector<double> ranksDiff(rankSize);
    thrust::device_ptr<double> newRankBegin(newRank);
    thrust::device_ptr<double> prevRankBegin(prevRank);
    thrust::device_ptr<double> prevRankEnd(prevRank + rankSize);
    thrust::transform(thrust::device,
        prevRankBegin, prevRankEnd, newRankBegin, ranksDiff.begin(), thrust::minus<double>()
    );

    return thrust::transform_reduce(thrust::device,
        ranksDiff.begin(), ranksDiff.end(), absolute_value(), 0.0, thrust::maximum<double>()
    );
}

/*****************************************************/
/* KERNEL: CALCULATE CID TOTAL OUTS STAKE            */
/*****************************************************/
__global__
void calculateCidTotalOutStake(
    uint64_t cidsSize,
    uint64_t *stakes,                                        /*array index - user index*/
    uint64_t *outLinksStartIndex, uint32_t *outLinksCount,   /*array index - cid index*/
    uint64_t *outLinksUsers,                                 /*all out links from all users*/
    /*returns*/ uint64_t *cidsTotalOutStakes                 /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        uint64_t totalOutStake = 0;
        for (uint64_t j = outLinksStartIndex[i]; j < outLinksStartIndex[i] + outLinksCount[i]; j++) {
           totalOutStake += stakes[outLinksUsers[j]];
        }
        cidsTotalOutStakes[i] = totalOutStake;
    }
}

/*********************************************************/
/* DEVICE: USER TO DIVIDE TWO uint64                     */
/*********************************************************/
__device__ __forceinline__
double ddiv_rn(uint64_t *a, uint64_t *b) {
    return __ddiv_rn(__ull2double_rn(*a), __ull2double_rn(*b));
}

/*****************************************************/
/* KERNEL: CALCULATE PERSONAL LINK NODE WEIGHT        */
/*****************************************************/
__global__
void calculateCyberlinksLocalWeights(
    uint64_t cidsSize,
    uint64_t *stakes,                                        /*array index - user index*/
    uint64_t *outLinksStartIndex, uint32_t *outLinksCount,   /*array index - cid index*/
    uint64_t *outLinksUsers,                                 /*all out links from all users*/
    uint64_t *cidsTotalOutStakes,                            /*array index - cid index*/
    uint64_t *cidsTotalInStakes,                             /*array index - cid index*/
    /*returns*/ double *cyberlinksLocalWeights                 /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        uint64_t oil = cidsTotalOutStakes[i] + cidsTotalInStakes[i]; 
        for (uint64_t j = outLinksStartIndex[i]; j < outLinksStartIndex[i] + outLinksCount[i]; j++) {
            double weight = ddiv_rn(&stakes[outLinksUsers[j]], &oil);
            cyberlinksLocalWeights[j] = weight;
        }
    }
}

/*****************************************************/
/* KERNEL: CALCULATE CIDS TOTAL ENTROPY              */
/*****************************************************/
__global__
void calculateNodeEntropy(
    uint64_t cidsSize,
    uint64_t *stakes,                                        /*array index - user index*/
    uint64_t *outLinksStartIndex, uint32_t *outLinksCount,   /*array index - cid index*/
    uint64_t *outLinksUsers,                                 /*all out links from all users*/
    uint64_t *cidsTotalOutStakes,                             /*array index - cid index*/
    uint64_t *cidsTotalInStakes,                             /*array index - cid index*/
    /*returns*/ double *nodesTotalEntropy               /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        double nodeLinksEntropy = 0;
        uint64_t oil = cidsTotalOutStakes[i] + cidsTotalInStakes[i]; 
        // uint64_t oil = cidsTotalOutStakes[i];
        for (uint64_t j = outLinksStartIndex[i]; j < outLinksStartIndex[i] + outLinksCount[i]; j++) {
           double weight = ddiv_rn(&stakes[outLinksUsers[j]], &oil);
           double logw = log2(weight);
           nodeLinksEntropy -= __dmul_rn(weight,logw);
        }
        nodesTotalEntropy[i] = nodeLinksEntropy;
    }
}

/*********************************************************/
/* KERNEL: MULTIPLY TWO ARRAYS                           */
/*********************************************************/
__global__
void mulArrays(
    uint64_t size,
    double *in1,
    double *in2,
    double *output
) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tx < size) output[tx] = __dmul_rn(in1[tx], in2[tx]);
}


// TODO: use for in out stakes
/*********************************************************/
/* KERNEL: SUM TWO ARRAYS                           */
/*********************************************************/
__global__ void sumArrays(
    uint64_t size,
    double *in1,
    double *in2,
    double *output
) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tx < size) output[tx] = __dadd_rn(in1[tx], in2[tx]);
}

// TODO: use for in out stakes
/*********************************************************/
/* KERNEL: CALCULATE SI                          */
/*********************************************************/
__global__ void calculate_SI(
    uint64_t size,
    uint64_t *out,
    uint64_t *in,
    double *d_si,
    double damping
) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tx < size) d_si[tx] = __dadd_rn(__dmul_rn(damping, __ull2double_rn(in[tx])), __dmul_rn(1-damping, __ull2double_rn(out[tx])));
}

/*****************************************************/
/* KERNEL: CALCULATE CIDS TOTAL ENTROPY              */
/*****************************************************/
__global__
void calculate_QJ(
    uint64_t cidsSize,
    uint64_t *outLinksStartIndex, uint32_t *outLinksCount,   /*array index - cid index*/
    uint64_t *inLinksOuts,  
    double *si,
    double damping,
    /*returns*/ double *qj
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        // double nodeLinksEntropy = 0;
        // uint64_t oil = cidsTotalOutStakes[i] + cidsTotalInStakes[i]; 
        // uint64_t oil = cidsTotalOutStakes[i];
        // double qj_node = 0;
        for (uint64_t j = outLinksStartIndex[i]; j < outLinksStartIndex[i] + outLinksCount[i]; j++) {
            qj[i] = __dadd_rn(qj[i],__dmul_rn(damping,si[inLinksOuts[j]]));

        //    double weight = ddiv_rn(&stakes[outLinksUsers[j]], &oil);
        //    double logw = log2(weight);
        //    nodeLinksEntropy -= __dmul_rn(weight,logw);
            // qj_node += __dmul_rn(damping,si[j]);
        }
        // qj[i] = qj_node;
    }
}

__global__
void calculate_ENT(
    uint64_t cidsSize,
    uint64_t *outLinksStartIndex, uint32_t *outLinksCount,   /*array index - cid index*/
    uint64_t *inLinksOuts,  
    double *si,
    double *qj,
    /*returns*/ double *ent
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        // double nodeLinksEntropy = 0;
        // uint64_t oil = cidsTotalOutStakes[i] + cidsTotalInStakes[i]; 
        // uint64_t oil = cidsTotalOutStakes[i];
        // double qj_node = 0;
        for (uint64_t j = outLinksStartIndex[i]; j < outLinksStartIndex[i] + outLinksCount[i]; j++) {
            double weight = __ddiv_rn(si[i],qj[inLinksOuts[j]]);
            double logw = log2(weight);
            ent[i] = __dadd_rn(ent[i],fabs(__dmul_rn(weight, logw)));

        //    double weight = ddiv_rn(&stakes[outLinksUsers[j]], &oil);
        //    double logw = log2(weight);
        //    nodeLinksEntropy -= __dmul_rn(weight,logw);
            // qj_node += __dmul_rn(damping,si[j]);
        }
        // qj[i] = qj_node;
    }
}

/*********************************************************/
/* KERNEL: CALCULATE COMPRESSED IN LINKS COUNT FOR CIDS  */
/*********************************************************/
__global__
void getCompressedInLinksCount(
    uint64_t cidsSize,
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount,                    /*array index - cid index*/
    uint64_t *inLinksOuts,                                                  /*all incoming links from all users*/
    /*returns*/ uint32_t *compressedInLinksCount                            /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {

        if(inLinksCount[i] == 0) {
            compressedInLinksCount[i]=0;
            continue;
        }

        uint32_t compressedLinksCount = 0;
        for(uint64_t j = inLinksStartIndex[i]; j < inLinksStartIndex[i]+inLinksCount[i]; j++) {
            if(j == inLinksStartIndex[i] || inLinksOuts[j] != inLinksOuts[j-1]) {
                compressedLinksCount++;
            }
        }
        compressedInLinksCount[i] = compressedLinksCount;
    }
}


/*********************************************************/
/* KERNEL: CALCULATE COMPRESSED IN LINKS                 */
/*********************************************************/
__global__
void getCompressedInLinks(
    uint64_t cidsSize,
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount, uint64_t *cidsTotalOutStakes,   /*array index - cid index*/
    uint64_t *inLinksOuts, uint64_t *inLinksUsers,                                       /*all incoming links from all users*/
    uint64_t *stakes,                                                                    /*array index - user index*/
    uint64_t *compressedInLinksStartIndex, uint32_t *compressedInLinksCount,             /*array index - cid index*/
    /*returns*/ CompressedInLink *compressedInLinks                                      /*all incoming compressed links*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {

        if(inLinksCount[i] == 0) {
            continue;
        }

        uint32_t compressedLinksIndex = compressedInLinksStartIndex[i];

        if(inLinksCount[i] == 1) {
            uint64_t oppositeCid = inLinksOuts[inLinksStartIndex[i]];
            uint64_t compressedLinkStake = stakes[inLinksUsers[inLinksStartIndex[i]]];
            double weight = ddiv_rn(&compressedLinkStake, &cidsTotalOutStakes[oppositeCid]);
            compressedInLinks[compressedLinksIndex] = CompressedInLink {oppositeCid, weight};
            continue;
        }

        uint64_t compressedLinkStake = 0;
        uint64_t lastLinkIndex = inLinksStartIndex[i] + inLinksCount[i] - 1;
        for(uint64_t j = inLinksStartIndex[i]; j < lastLinkIndex + 1; j++) {

            compressedLinkStake += stakes[inLinksUsers[j]];
            if(j == lastLinkIndex || inLinksOuts[j] != inLinksOuts[j+1]) {
                uint64_t oppositeCid = inLinksOuts[j];
                double weight = ddiv_rn(&compressedLinkStake, &cidsTotalOutStakes[oppositeCid]);
                compressedInLinks[compressedLinksIndex] = CompressedInLink {oppositeCid, weight};
                compressedLinksIndex++;
                compressedLinkStake=0;
            }
        }
    }
}

__global__
void calculateKarma(
    uint64_t cidsSize,
    uint64_t *outLinksStartIndex, uint32_t *outLinksCount,
    uint64_t *outLinksUsers,      
    double *cyberlinksLocalWeights,
    double *light,
    /*returns*/ double *karma
) {
    for (uint64_t i = 0; i < cidsSize; i++) {          
        for (uint64_t j = outLinksStartIndex[i]; j < outLinksStartIndex[i] + outLinksCount[i]; j++) {
            karma[outLinksUsers[j]] += light[i]*cyberlinksLocalWeights[j];
        }
    }
}

/************************************************************/
/* HOST: CALCULATE COMPRESSED IN LINKS START INDEXES        */
/************************************************************/
/* SEQUENTIAL LOGIC -> CALCULATE ON CPU                     */
/* RETURNS TOTAL COMPRESSED LINKS SIZE                      */
/************************************************************/
__host__
uint64_t getLinksStartIndex(
    uint64_t cidsSize,
    uint32_t *linksCount,                   /*array index - cid index*/
    /*returns*/ uint64_t *linksStartIndex   /*array index - cid index*/
) {

    uint64_t index = 0;
    for (uint64_t i = 0; i < cidsSize; i++) {
        linksStartIndex[i] = index;
        index += linksCount[i];
    }
    return index;
}

void swap(double* &a, double* &b){
  double *temp = a;
  a = b;
  b = temp;
}

void printSize(size_t usageOffset) {
	size_t free = 0, total = 0;
	hipMemGetInfo(&free, &total);
	fprintf(stderr, "-[GPU]: Free: %.2fMB\tUsed: %.2fMB\n", free / 1048576.0f, (total - usageOffset - free) / 1048576.0f);
}

extern "C" {

    void calculate_rank(
        uint64_t *stakes, uint64_t stakesSize,                    /* User stakes and corresponding array size */
        uint64_t cidsSize, uint64_t linksSize,                    /* Cids count */
        uint32_t *inLinksCount, uint32_t *outLinksCount,          /* array index - cid index*/
        uint64_t *outLinksIns,
        uint64_t *inLinksOuts, uint64_t *inLinksUsers,            /*all incoming links from all users*/
        uint64_t *outLinksUsers,                                  /*all outgoing links from all users*/
        double *rank,                                             /* array index - cid index*/
        double dampingFactor,                                     /* value of damping factor*/
        double tolerance,                                         /* value of needed tolerance */
        double *entropy,                                          /* array index - cid index*/
        double *light,                                            /* array index - cid index*/
        double *karma                                             /* array index - account index*/
    ) {

        // setbuf(stdout, NULL);
        int CUDA_BLOCKS_NUMBER = (cidsSize + CUDA_THREAD_BLOCK_SIZE - 1) / CUDA_THREAD_BLOCK_SIZE;

        size_t freeStart = 0, totalStart = 0, usageOffset = 0;
        hipMemGetInfo(&freeStart, &totalStart);
        usageOffset = totalStart - freeStart;
        fprintf(stderr, "[GPU]: Usage Offset: %.2fMB\n", usageOffset / 1048576.0f);

        // STEP0: Calculate compressed in links start indexes
        /*-------------------------------------------------------------------*/
        // calculated on cpu
        printf("STEP0: Calculate compressed in links start indexes\n");

        uint64_t *inLinksStartIndex = (uint64_t*) malloc(cidsSize*sizeof(uint64_t));
        uint64_t *outLinksStartIndex = (uint64_t*) malloc(cidsSize*sizeof(uint64_t));
        getLinksStartIndex(cidsSize, inLinksCount, inLinksStartIndex);
        getLinksStartIndex(cidsSize, outLinksCount, outLinksStartIndex);
        
        printSize(usageOffset);

        // STEP1: Calculate for each cid total stake by out links
        /*-------------------------------------------------------------------*/
        printf("STEP1: Calculate for each cid total stake by out links\n");
        
        uint64_t *d_outLinksStartIndex;
        uint32_t *d_outLinksCount;
        uint64_t *d_outLinksUsers;
        uint64_t *d_stakes;  // will be used to calculated links weights, should be freed before rank iterations
        uint64_t *d_cidsTotalOutStakes; // will be used to calculated links weights, should be freed before rank iterations

        hipMalloc(&d_outLinksStartIndex, cidsSize*sizeof(uint64_t));
        hipMalloc(&d_outLinksCount,      cidsSize*sizeof(uint32_t));
        hipMalloc(&d_outLinksUsers,     linksSize*sizeof(uint64_t));
        hipMalloc(&d_stakes,           stakesSize*sizeof(uint64_t));
        hipMalloc(&d_cidsTotalOutStakes, cidsSize*sizeof(uint64_t));   //calculated

        hipMemcpy(d_outLinksStartIndex, outLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_outLinksCount,      outLinksCount,      cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_outLinksUsers,      outLinksUsers,     linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_stakes,             stakes,           stakesSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        calculateCidTotalOutStake<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_stakes, d_outLinksStartIndex,
            d_outLinksCount, d_outLinksUsers, d_cidsTotalOutStakes
        );

        printSize(usageOffset);

        // DEV ENTROPY (in+out stake)
        /*-------------------------------------------------------------------*/
        printf("DEV ENTROPY 222- IN STAKE\n");

        uint64_t *d_inLinksStartIndex0;
        uint32_t *d_inLinksCount0;
        uint64_t *d_inLinksUsers0;
        uint64_t *d_cidsTotalInStakes; // will be used to calculated links weights, should be freed before rank iterations

        hipMalloc(&d_inLinksStartIndex0, cidsSize*sizeof(uint64_t));
        hipMalloc(&d_inLinksCount0,      cidsSize*sizeof(uint32_t));
        hipMalloc(&d_inLinksUsers0,      linksSize*sizeof(uint64_t));
        hipMalloc(&d_cidsTotalInStakes, cidsSize*sizeof(uint64_t));   //calculated
        
        hipMemcpy(d_inLinksStartIndex0, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_inLinksCount0,      inLinksCount,      cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_inLinksUsers0,      inLinksUsers,      linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        calculateCidTotalOutStake<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_stakes, d_inLinksStartIndex0,
            d_inLinksCount0, d_inLinksUsers0, d_cidsTotalInStakes
        );

        // hipFree(d_inLinksStartIndex0);
        // hipFree(d_inLinksCount0);
        // hipFree(d_inLinksUsers0);

        // thrust::device_ptr<uint64_t> outP(d_cidsTotalOutStakes);
        // thrust::device_ptr<uint64_t> inP(d_cidsTotalInStakes);
        // for(uint64_t i = 0; i < 21; i++) {
        //     printf("[%d] = %d | %d\n",i,(uint64_t)*(outP+i), (uint64_t)*(inP+i));
        // }
        printSize(usageOffset);


        double *d_si;
        hipMalloc(&d_si, cidsSize*sizeof(double));
        hipMemcpy(d_si, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        calculate_SI<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_cidsTotalOutStakes, d_cidsTotalInStakes, d_si, dampingFactor);
        thrust::device_ptr<double> SI(d_si);
        for(uint64_t i = 0; i < 21; i++) {
            printf("[%d] = %f\n",i,(double)*(SI+i));
        }

        printf("DEV ENTROPY 222- ENTROPY OUT\n");

        double *d_entropy_out;
        hipMalloc(&d_entropy_out, cidsSize*sizeof(double));
        hipMemcpy(d_entropy_out, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        calculateNodeEntropy<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_stakes, d_outLinksStartIndex,
            d_outLinksCount, d_outLinksUsers, d_cidsTotalOutStakes, d_cidsTotalInStakes, d_entropy_out
        );
        // hipMemcpy(entropy, d_entropy, cidsSize * sizeof(double), hipMemcpyDeviceToHost);
        
        printSize(usageOffset);
        
        /*-----------*/

        printf("DEV ENTROPY - ENTROPY IN\n");

        double *d_entropy_in;
        hipMalloc(&d_entropy_in, cidsSize*sizeof(double));
        hipMemcpy(d_entropy_in, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        calculateNodeEntropy<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_stakes, d_inLinksStartIndex0,
            d_inLinksCount0, d_inLinksUsers0, d_cidsTotalOutStakes, d_cidsTotalInStakes, d_entropy_in
        );
        // hipMemcpy(entropy, d_entropy, cidsSize * sizeof(double), hipMemcpyDeviceToHost);
        
        // TODO Refactor steps, optimize allocation
        // hipFree(d_inLinksStartIndex0);
        // hipFree(d_inLinksCount0);
        // hipFree(d_inLinksUsers0);

        printSize(usageOffset);

                
        /*-----------*/
        printf("SUM ENTROPY - IN+OUT\n");

        double *d_entropy;
        hipMalloc(&d_entropy, cidsSize*sizeof(double));
        hipMemcpy(d_entropy, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        sumArrays<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_entropy_out, d_entropy_in, d_entropy
        );

        hipFree(d_entropy_out);
        hipFree(d_entropy_in);

        printSize(usageOffset);
        /*-----------*/

        printf("LOCAL WEIGHTS\n");

        double *d_cyberlinksLocalWeights;
        hipMalloc(&d_cyberlinksLocalWeights, linksSize*sizeof(double));
        
        calculateCyberlinksLocalWeights<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_stakes, d_outLinksStartIndex,
            d_outLinksCount, d_outLinksUsers, d_cidsTotalOutStakes, d_cidsTotalInStakes, d_cyberlinksLocalWeights
        );

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/

        // hipFree(d_outLinksStartIndex);
        // hipFree(d_outLinksCount);
        // hipFree(d_outLinksUsers);
        /*-------------------------------------------------------------------*/



        // STEP2: Calculate compressed in links count
        /*-------------------------------------------------------------------*/
        printf("STEP2: Calculate compressed in links count\n");

        uint64_t *d_inLinksStartIndex;
        uint32_t *d_inLinksCount;
        uint64_t *d_inLinksOuts;
        uint32_t *d_compressedInLinksCount;

        // free all before rank iterations
        hipMalloc(&d_inLinksStartIndex,      cidsSize*sizeof(uint64_t));
        hipMalloc(&d_inLinksCount,           cidsSize*sizeof(uint32_t));
        hipMalloc(&d_inLinksOuts,           linksSize*sizeof(uint64_t));
        hipMalloc(&d_compressedInLinksCount, cidsSize*sizeof(uint32_t));   //calculated

        hipMemcpy(d_inLinksStartIndex, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_inLinksCount,      inLinksCount,      cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_inLinksOuts,       inLinksOuts,      linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        getCompressedInLinksCount<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_inLinksStartIndex, d_inLinksCount, d_inLinksOuts, d_compressedInLinksCount
        );
        printSize(usageOffset);
        /*-------------------------------------------------------------------*/

        printf("-______________in____________\n");

        double *d_qj_in;
        hipMalloc(&d_qj_in, cidsSize*sizeof(double));
        hipMemset(d_qj_in, 0, cidsSize*sizeof(double));
        // hipMemcpy(d_qj_in, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        calculate_QJ<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_inLinksStartIndex,
            d_inLinksCount, d_inLinksOuts, d_si, 0.8f, d_qj_in);
        
        thrust::device_ptr<double> QJ_IN(d_qj_in);
        for(uint64_t i = 0; i < 21; i++) {
            printf("[%d] = %f\n",i,(double)*(QJ_IN+i));
        }

        printf("-_______________out__________\n");

        uint64_t *d_outLinksIns;
        hipMalloc(&d_outLinksIns,linksSize*sizeof(uint64_t));
        hipMemcpy(d_outLinksIns,outLinksIns,linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        double *d_qj_out;
        hipMalloc(&d_qj_out, cidsSize*sizeof(double));
        hipMemset(d_qj_out, 0, cidsSize*sizeof(double));
        // hipMemcpy(d_qj_out, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        calculate_QJ<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_outLinksStartIndex,
            d_outLinksCount, d_outLinksIns, d_si, 0.2f, d_qj_out);
        
        thrust::device_ptr<double> QJ_OUT(d_qj_out);
        for(uint64_t i = 0; i < 21; i++) {
            printf("[%d] = %f\n",i,(double)*(QJ_OUT+i));
        }

        printf("-_____________sum____________\n");

        double *d_qj_sum;
        hipMalloc(&d_qj_sum, cidsSize*sizeof(double));
        hipMemcpy(d_qj_sum, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        sumArrays<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_qj_out, d_qj_in, d_qj_sum
        );
        thrust::device_ptr<double> QJ(d_qj_sum);
        for(uint64_t i = 0; i < 21; i++) {
            printf("[%d] = %f\n",i,(double)*(QJ+i));
        }

                printf("-______________in-ent____________\n");

        double *d_ent_chi;
        hipMalloc(&d_ent_chi, cidsSize*sizeof(double));
        hipMemset(d_ent_chi, 0, cidsSize*sizeof(double));
        // hipMemcpy(d_qj_in, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        calculate_ENT<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_inLinksStartIndex,
            d_inLinksCount, d_inLinksOuts, d_si, d_qj_sum, d_ent_chi);

        calculate_ENT<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
        cidsSize, d_outLinksStartIndex,
        d_outLinksCount, d_outLinksIns, d_si, d_qj_sum, d_ent_chi);
        
        thrust::device_ptr<double> ENT(d_ent_chi);
        for(uint64_t i = 0; i < 21; i++) {
            printf("[%d] = %f\n",i,(double)*(ENT+i));
        }

        hipFree(d_si);
        hipFree(d_qj_in);
        hipFree(d_qj_out);
        hipFree(d_qj_sum);
        hipFree(d_ent_chi);


        // STEP3: Calculate compressed in links start indexes
        /*-------------------------------------------------------------------*/
        printf("STEP3: Calculate compressed in links start indexes\n");

        uint32_t *compressedInLinksCount = (uint32_t*) malloc(cidsSize*sizeof(uint32_t));
        uint64_t *compressedInLinksStartIndex = (uint64_t*) malloc(cidsSize*sizeof(uint64_t));
        hipMemcpy(compressedInLinksCount, d_compressedInLinksCount, cidsSize * sizeof(uint32_t), hipMemcpyDeviceToHost);

        // calculated on cpu
        uint64_t compressedInLinksSize = getLinksStartIndex(
            cidsSize, compressedInLinksCount, compressedInLinksStartIndex
        );

        uint64_t *d_compressedInLinksStartIndex;
        hipMalloc(&d_compressedInLinksStartIndex, cidsSize*sizeof(uint64_t));
        hipMemcpy(d_compressedInLinksStartIndex, compressedInLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        free(compressedInLinksStartIndex);

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/

        // STEP4: Calculate compressed in links
        /*-------------------------------------------------------------------*/
        printf("STEP4: Calculate compressed in links\n");

        uint64_t *d_inLinksUsers;
        CompressedInLink *d_compressedInLinks; //calculated

        hipMalloc(&d_inLinksUsers,                   linksSize*sizeof(uint64_t));
        hipMalloc(&d_compressedInLinks,  compressedInLinksSize*sizeof(CompressedInLink));
        hipMemcpy(d_inLinksUsers, inLinksUsers,      linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        getCompressedInLinks<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize,
            d_inLinksStartIndex, d_inLinksCount, d_cidsTotalOutStakes,
            d_inLinksOuts, d_inLinksUsers, d_stakes,
            d_compressedInLinksStartIndex, d_compressedInLinksCount,
            d_compressedInLinks
        );

        hipFree(d_inLinksUsers);
        hipFree(d_inLinksStartIndex);
        hipFree(d_inLinksCount);
        hipFree(d_inLinksOuts);
        hipFree(d_stakes);
        hipFree(d_cidsTotalOutStakes);
        hipFree(d_cidsTotalInStakes);

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/



        // STEP5: Calculate dangling nodes rank, and default rank
        /*-------------------------------------------------------------------*/
        printf("STEP5: Calculate dangling nodes rank, and default rank\n");

        double defaultRank = (1.0 - dampingFactor) / cidsSize;
        uint64_t danglingNodesSize = 0;
        for(uint64_t i=0; i< cidsSize; i++){
            rank[i] = defaultRank;
            if(inLinksCount[i] == 0) {
                danglingNodesSize++;
            }
        }

        double innerProductOverSize = defaultRank * ((double) danglingNodesSize / (double)cidsSize);
        double defaultRankWithCorrection = (dampingFactor * innerProductOverSize) + defaultRank; //fma point

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/




        // STEP6: Calculate Rank
        /*-------------------------------------------------------------------*/
        printf("STEP6: Calculate Rank\n");

        double *d_rank, *d_prevRank;

        hipMalloc(&d_rank, cidsSize*sizeof(double));
        hipMalloc(&d_prevRank, cidsSize*sizeof(double));

        hipMemcpy(d_rank,     rank, cidsSize*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_prevRank, rank, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        int steps = 0;
        double change = tolerance + 1.0;
        while(change > tolerance) {
            swap(d_rank, d_prevRank);
            steps++;
        	run_rank_iteration<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
                d_compressedInLinks,
                d_prevRank, d_rank, cidsSize,
                d_compressedInLinksStartIndex, d_compressedInLinksCount,
                defaultRankWithCorrection, dampingFactor
        	);
        	change = find_max_ranks_diff(d_prevRank, d_rank, cidsSize);
        	hipDeviceSynchronize();
        }

        hipMemcpy(rank, d_rank, cidsSize * sizeof(double), hipMemcpyDeviceToHost);
        
        printSize(usageOffset);
        /*-------------------------------------------------------------------*/
        printf("STEP7: Calculate Light\n");

        double *d_light;
        hipMalloc(&d_light, cidsSize*sizeof(double));
        hipMemcpy(d_light, light, cidsSize*sizeof(double), hipMemcpyHostToDevice);
        mulArrays<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize,d_rank,d_entropy,d_light
        );
        hipMemcpy(light, d_light, cidsSize * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_entropy);
        hipFree(d_rank);
        hipFree(d_prevRank);
        hipFree(d_compressedInLinksStartIndex);
        hipFree(d_compressedInLinksCount);
        hipFree(d_compressedInLinks);

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/
        printf("STEP8: Calculate Karma\n");

        double *d_karma;
        hipMalloc(&d_karma, stakesSize*sizeof(double));
        hipMemcpy(d_karma, karma, stakesSize*sizeof(double), hipMemcpyHostToDevice);
        calculateKarma<<<1,1>>>(
            cidsSize,
            d_outLinksStartIndex,
            d_outLinksCount,
            d_outLinksUsers,
            d_cyberlinksLocalWeights,
            d_light,
            d_karma
        );
        hipMemcpy(karma, d_karma, stakesSize * sizeof(double), hipMemcpyDeviceToHost);
        printSize(usageOffset);
        /*-----------------*/
        printf("STEP9: Cleaning\n");

        hipFree(d_outLinksStartIndex);
        hipFree(d_outLinksCount);
        hipFree(d_outLinksUsers);

        hipFree(d_light);
        hipFree(d_karma);

        hipFree(d_cyberlinksLocalWeights);

        printSize(usageOffset);
    }
};
