#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "types.h"

static void HandleError(
    hipError_t err,
    const char *file,
    int line
) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

const int CUDA_THREAD_BLOCK_SIZE = 256;

/*****************************************************/
/* KERNEL: RUN SINGLE RANK ITERATION                 */
/*****************************************************/
/* All in links used here are compressed in links    */
/*****************************************************/
__global__
void run_rank_iteration(
    CompressedInLink *inLinks,                            /* all compressed in links */
    double *prevRank, double *rank, uint64_t rankSize,    /* array index - cid index */
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount,  /* array index - cid index */
    double defaultRankWithCorrection,                     /* default rank + inner product correction */
    double dampingFactor
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < rankSize; i += stride) {

        if(inLinksCount[i] == 0) {
            continue;
        }

        double ksum = 0;
        for (uint64_t j = inLinksStartIndex[i]; j < inLinksStartIndex[i] + inLinksCount[i]; j++) {
           if (inLinks[j].weight == 0) {continue;}
           ksum = prevRank[inLinks[j].fromIndex] * inLinks[j].weight + ksum;
        }
        rank[i] = ksum * dampingFactor + defaultRankWithCorrection;
    }
}


/*****************************************************/
/* KERNEL: DOUBLE ABS FUNCTOR                        */
/*****************************************************/
/* Return absolute value for double                  */
/*****************************************************/
struct absolute_value {
  __device__ double operator()(const double &x) const {
    return x < 0.0 ? -x : x;
  }
};


/*****************************************************/
/* HOST: FINDS MAXIMUM RANKS DIFFERENCE              */
/*****************************************************/
/* Finds maximum rank difference for single element  */
/*                                                   */
/*****************************************************/
double find_max_ranks_diff(double *prevRank, double *newRank, uint64_t rankSize) {

    thrust::device_vector<double> ranksDiff(rankSize);
    thrust::device_ptr<double> newRankBegin(newRank);
    thrust::device_ptr<double> prevRankBegin(prevRank);
    thrust::device_ptr<double> prevRankEnd(prevRank + rankSize);
    thrust::transform(thrust::device,
        prevRankBegin, prevRankEnd, newRankBegin, ranksDiff.begin(), thrust::minus<double>()
    );

    return thrust::transform_reduce(thrust::device,
        ranksDiff.begin(), ranksDiff.end(), absolute_value(), 0.0, thrust::maximum<double>()
    );
}

/*******************************************************/
/* KERNEL: CALCULATE PARTICLE STAKE BY IN OR OUT LINKS */
/*******************************************************/
__global__
void get_particle_stake_by_links(
    uint64_t cidsSize,
    uint64_t *stakes,                                /*array index - user index*/
    uint64_t *linksStartIndex, uint32_t *linksCount, /*array index - cid index*/
    uint64_t *linksUsers,                            /*all links from all users*/
    /*returns*/ uint64_t *cidsTotalOutStakes         /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        uint64_t totalOutStake = 0;
        for (uint64_t j = linksStartIndex[i]; j < linksStartIndex[i] + linksCount[i]; j++) {
           totalOutStake += stakes[linksUsers[j]];
        }
        cidsTotalOutStakes[i] = totalOutStake;
    }
}

/*********************************************************/
/* DEVICE: USER TO DIVIDE TWO uint64                     */
/*********************************************************/
__device__ __forceinline__
double ddiv_rn(uint64_t *a, uint64_t *b) {
    return __ddiv_rn(__ull2double_rn(*a), __ull2double_rn(*b));
}

/*****************************************************/
/* KERNEL: CALCULATE CYBERLINKS WEIGHTS BY STAKE     */
/*****************************************************/
__global__
void get_cyberlinks_weight_by_stake(
    uint64_t cidsSize,
    uint64_t *stakes,                                /*array index - user index*/
    uint64_t *linksStartIndex, uint32_t *linksCount, /*array index - cid index*/
    uint64_t *linksUsers,                            /*all out links from all users*/
    uint64_t *cidsTotalStakes,                       /*array index - cid index*/
    /*returns*/ double *cyberlinksLocalWeights       /*array index - links index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        uint64_t stake = cidsTotalStakes[i]; 
        for (uint64_t j = linksStartIndex[i]; j < linksStartIndex[i] + linksCount[i]; j++) {
            double weight = ddiv_rn(&stakes[linksUsers[j]], &stake);
            cyberlinksLocalWeights[j] = weight;
        }
    }
}

/*********************************************************/
/* KERNEL: MULTIPLY TWO ARRAYS                           */
/*********************************************************/
__global__
void multiply_arrays(
    uint64_t size,
    double   *a,
    double   *b,
    double   *output
) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tx < size) output[tx] = __dmul_rn(a[tx], b[tx]);
}

/*************************************************************************/
/* KERNEL: CALCULATE PARTICLE TOTAL STAKE TRANSORMED WITH DAMPING FACTOR */
/*************************************************************************/
__global__
void get_stake_with_damping(
    uint64_t size,
    uint64_t *outStake,
    uint64_t *inStake,
    double   *swd,
    double   damping
) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tx < size) swd[tx] = __dadd_rn(
        __dmul_rn(damping, __ull2double_rn(inStake[tx])),
        __dmul_rn(1-damping, __ull2double_rn(outStake[tx]))
    );
}

/******************************************************************************************/
/* KERNEL: CALCULATE SUM OF ADJACENT PARTICLES STAKE WITH DAMPING BY IN OR OUT CYBERLINKS */
/******************************************************************************************/
__global__
void sum_stake_with_damping_by_links(
    uint64_t cidsSize,
    uint64_t *linksStartIndex, uint32_t *linksCount, /*array index - cid index*/
    uint64_t *linksOuts, // linksIns                 /*all incoming or outgoing links from all users*/
    double *swd,                                     /*array index - cid index*/
    double damping,
    /*returns*/ double *sumswd                       /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        for (uint64_t j = linksStartIndex[i]; j < linksStartIndex[i] + linksCount[i]; j++) {
            sumswd[i] = __dadd_rn(sumswd[i], __dmul_rn(damping, swd[linksOuts[j]]));
        }
    }
}

/******************************************************************/
/* KERNEL: CALCULATE ENTROPY BY IN OR OUT CYBERLINKS FOR PARTICLE */
/******************************************************************/
__global__
void calculate_entropy_by_links(
    uint64_t cidsSize,
    uint64_t *linksStartIndex, uint32_t *linksCount, /*array index - cid index*/
    uint64_t *linksOuts, // linksIns                 /*all incoming or outgoing links from all users*/
    double *swd,                                     /*array index - cid index*/
    double *d_sumswd,                                /*array index - cid index*/
    /*returns*/ double *entropy                      /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        for (uint64_t j = linksStartIndex[i]; j < linksStartIndex[i] + linksCount[i]; j++) {
            if (swd[i] == 0 || d_sumswd[linksOuts[j]] == 0) { continue; }
            double weight = __ddiv_rn(swd[i], d_sumswd[linksOuts[j]]);
            // if (isnan(weight)) { continue; }
            double logw = log2(weight);
            entropy[i] = __dadd_rn(entropy[i], fabs(__dmul_rn(weight, logw)));
        }
    }
}

/*********************************************************/
/* KERNEL: CALCULATE COMPRESSED IN LINKS COUNT FOR CIDS  */
/*********************************************************/
__global__
void get_compressed_in_links_count(
    uint64_t cidsSize,
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount, /*array index - cid index*/
    uint64_t *inLinksOuts,                               /*all incoming links from all users*/
    /*returns*/ uint32_t *compressedInLinksCount         /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {

        if(inLinksCount[i] == 0) {
            compressedInLinksCount[i]=0;
            continue;
        }

        uint32_t compressedLinksCount = 0;
        for(uint64_t j = inLinksStartIndex[i]; j < inLinksStartIndex[i]+inLinksCount[i]; j++) {
            if(j == inLinksStartIndex[i] || inLinksOuts[j] != inLinksOuts[j-1]) {
                compressedLinksCount++;
            }
        }
        compressedInLinksCount[i] = compressedLinksCount;
    }
}


/*********************************************************/
/* KERNEL: CALCULATE COMPRESSED IN LINKS                 */
/*********************************************************/
__global__
void get_compressed_in_links(
    uint64_t cidsSize,
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount, uint64_t *cidsTotalOutStakes, /*array index - cid index*/
    uint64_t *inLinksOuts, uint64_t *inLinksUsers,                                     /*all incoming links from all users*/
    uint64_t *stakes,                                                                  /*array index - user index*/
    uint64_t *compressedInLinksStartIndex, uint32_t *compressedInLinksCount,           /*array index - cid index*/
    /*returns*/ CompressedInLink *compressedInLinks                                    /*all incoming compressed links*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {

        if(inLinksCount[i] == 0) {
            continue;
        }

        uint32_t compressedLinksIndex = compressedInLinksStartIndex[i];

        if(inLinksCount[i] == 1) {
            uint64_t oppositeCid = inLinksOuts[inLinksStartIndex[i]];
            uint64_t compressedLinkStake = stakes[inLinksUsers[inLinksStartIndex[i]]];
            double weight = ddiv_rn(&compressedLinkStake, &cidsTotalOutStakes[oppositeCid]);
            if (isnan(weight)) { weight = 0; }
            compressedInLinks[compressedLinksIndex] = CompressedInLink {oppositeCid, weight};
            continue;
        }

        uint64_t compressedLinkStake = 0;
        uint64_t lastLinkIndex = inLinksStartIndex[i] + inLinksCount[i] - 1;
        for(uint64_t j = inLinksStartIndex[i]; j < lastLinkIndex + 1; j++) {

            compressedLinkStake += stakes[inLinksUsers[j]];
            if(j == lastLinkIndex || inLinksOuts[j] != inLinksOuts[j+1]) {
                uint64_t oppositeCid = inLinksOuts[j];
                double weight = ddiv_rn(&compressedLinkStake, &cidsTotalOutStakes[oppositeCid]);
                if (isnan(weight)) { weight = 0; }
                compressedInLinks[compressedLinksIndex] = CompressedInLink {oppositeCid, weight};
                compressedLinksIndex++;
                compressedLinkStake=0;
            }
        }
    }
}

/************************************************************/
/* HOST: CALCULATE KARMA                                    */
/************************************************************/
/* SEQUENTIAL LOGIC -> CALCULATE ON CPU                     */
/* RETURNS KARMA FOR ALL ACCOUNTS                           */
/************************************************************/
// __host__
__global__
void calculate_karma(
    uint64_t cidsSize,
    uint64_t *outLinksStartIndex, uint32_t *outLinksCount,
    uint64_t *outLinksUsers,      
    double   *cyberlinksLocalWeights,
    double   *light,
    /*returns*/ double *karma
) {
    for (uint64_t i = 0; i < cidsSize; i++) {          
        for (uint64_t j = outLinksStartIndex[i]; j < outLinksStartIndex[i] + outLinksCount[i]; j++) {
            // karma[outLinksUsers[j]] += light[i]*cyberlinksLocalWeights[j];
            karma[outLinksUsers[j]] = __dadd_rn(karma[outLinksUsers[j]], __dmul_rn(light[i],cyberlinksLocalWeights[j]));
            // printf("[%d][%d] = %lf | %lf\n", i, j, karma[outLinksUsers[j]], karma[outLinksUsers[j]]);
        }
    }
}

/************************************************************/
/* HOST: CALCULATE COMPRESSED IN LINKS START INDEXES        */
/************************************************************/
/* SEQUENTIAL LOGIC -> CALCULATE ON CPU                     */
/* RETURNS TOTAL COMPRESSED LINKS SIZE                      */
/************************************************************/
__host__
uint64_t get_links_start_index(
    uint64_t cidsSize,
    uint32_t *linksCount,                   /*array index - cid index*/
    /*returns*/ uint64_t *linksStartIndex   /*array index - cid index*/
) {

    uint64_t index = 0;
    for (uint64_t i = 0; i < cidsSize; i++) {
        linksStartIndex[i] = index;
        index += linksCount[i];
    }
    return index;
}

void swap(double* &a, double* &b){
  double *temp = a;
  a = b;
  b = temp;
}

void printSize(size_t usageOffset) {
	size_t free = 0, total = 0;
	hipMemGetInfo(&free, &total);
	fprintf(stderr, "-[GPU]: Free: %.2fMB\tUsed: %.2fMB\n", free / 1048576.0f, (total - usageOffset - free) / 1048576.0f);
}

extern "C" {

    void calculate_rank(
        uint64_t *stakes, uint64_t stakesSize,                    /* User stakes and corresponding array size */
        uint64_t cidsSize, uint64_t linksSize,                    /* Cids count */
        uint32_t *inLinksCount, uint32_t *outLinksCount,          /* array index - cid index*/
        uint64_t *outLinksIns,
        uint64_t *inLinksOuts, uint64_t *inLinksUsers,            /*all incoming links from all users*/
        uint64_t *outLinksUsers,                                  /*all outgoing links from all users*/
        double *rank,                                             /* array index - cid index*/
        double dampingFactor,                                     /* value of damping factor*/
        double tolerance,                                         /* value of needed tolerance */
        double *entropy,                                          /* array index - cid index*/
        double *light,                                            /* array index - cid index*/
        double *karma                                             /* array index - account index*/
    ) {

        // setbuf(stdout, NULL);
        int CUDA_BLOCKS_NUMBER = (cidsSize + CUDA_THREAD_BLOCK_SIZE - 1) / CUDA_THREAD_BLOCK_SIZE;

        size_t freeStart = 0, totalStart = 0, usageOffset = 0;
        hipMemGetInfo(&freeStart, &totalStart);
        usageOffset = totalStart - freeStart;
        fprintf(stderr, "[GPU]: Usage Offset: %.2fMB\n", usageOffset / 1048576.0f);

        // STEP0: Calculate compressed in/out links start indexes
        /*-------------------------------------------------------------------*/
        // calculated on CPU
        printf("STEP0: Calculate compressed in/out links start indexes\n");

        uint64_t *inLinksStartIndex = (uint64_t*) malloc(cidsSize*sizeof(uint64_t));
        uint64_t *outLinksStartIndex = (uint64_t*) malloc(cidsSize*sizeof(uint64_t));
        get_links_start_index(cidsSize, inLinksCount, inLinksStartIndex);
        get_links_start_index(cidsSize, outLinksCount, outLinksStartIndex);
        
        printSize(usageOffset);
        /*-------------------------------------------------------------------*/


        // STEP1.1: Calculate for each particle stake by OUT cyberlinks
        /*-------------------------------------------------------------------*/
        printf("STEP1.1: Calculate for each particle stake by OUT cyberlinks\n");
        
        uint64_t *d_outLinksStartIndex;
        uint32_t *d_outLinksCount;
        uint64_t *d_outLinksUsers;
        uint64_t *d_stakes;             // will be used to calculated links weights, should be freed before rank iterations
        uint64_t *d_cidsTotalOutStakes; // will be used to calculated links weights, should be freed before rank iterations

        hipMalloc(&d_outLinksStartIndex, cidsSize*sizeof(uint64_t));
        hipMalloc(&d_outLinksCount,      cidsSize*sizeof(uint32_t));
        hipMalloc(&d_outLinksUsers,     linksSize*sizeof(uint64_t));
        hipMalloc(&d_stakes,           stakesSize*sizeof(uint64_t));
        hipMalloc(&d_cidsTotalOutStakes, cidsSize*sizeof(uint64_t)); //calculated

        hipMemcpy(d_outLinksStartIndex, outLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_outLinksCount,      outLinksCount,      cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_outLinksUsers,      outLinksUsers,     linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_stakes,             stakes,           stakesSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        get_particle_stake_by_links<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_stakes, d_outLinksStartIndex,
            d_outLinksCount, d_outLinksUsers, d_cidsTotalOutStakes
        );

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/


        // STEP1.2: Calculate for each particle total stake by IN links
        /*-------------------------------------------------------------------*/
        printf("STEP1.2: Calculate for each particle stake by IN links\n");

        uint64_t *d_inLinksStartIndex;
        uint32_t *d_inLinksCount;
        uint64_t *d_inLinksUsers;
        uint64_t *d_cidsTotalInStakes; // will be used to calculated links weights, should be freed before rank iterations

        hipMalloc(&d_inLinksStartIndex, cidsSize*sizeof(uint64_t));
        hipMalloc(&d_inLinksCount,      cidsSize*sizeof(uint32_t));
        hipMalloc(&d_inLinksUsers,      linksSize*sizeof(uint64_t));
        hipMalloc(&d_cidsTotalInStakes, cidsSize*sizeof(uint64_t));   //calculated
        
        hipMemcpy(d_inLinksStartIndex, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_inLinksCount,      inLinksCount,      cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_inLinksUsers,      inLinksUsers,      linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        get_particle_stake_by_links<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_stakes, d_inLinksStartIndex,
            d_inLinksCount, d_inLinksUsers, d_cidsTotalInStakes
        );

        // thrust::device_ptr<uint64_t> outP(d_cidsTotalOutStakes);
        // thrust::device_ptr<uint64_t> inP(d_cidsTotalInStakes);
        // for(uint64_t i = 0; i < 21; i++) {
        //     printf("[%d] = %d | %d\n",i,(uint64_t)*(outP+i), (uint64_t)*(inP+i));
        // }
        printSize(usageOffset);
       /*-------------------------------------------------------------------*/

        
        // STEP1.3: Calculate Stake With Damping
        /*-------------------------------------------------------------------*/
        printf("STEP1.3: Calculate Stake With Damping\n");

        double *d_swd;
        hipMalloc(&d_swd, cidsSize*sizeof(double));
        hipMemcpy(d_swd, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        get_stake_with_damping<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_cidsTotalOutStakes, d_cidsTotalInStakes, d_swd, dampingFactor);
        // thrust::device_ptr<double> SI(d_swd);
        // for(uint64_t i = 0; i < 21; i++) {
        //     printf("[%d] = %f\n",i,(double)*(SI+i));
        // }
        hipFree(d_cidsTotalInStakes);
        printSize(usageOffset);
        /*-------------------------------------------------------------------*/


        // STEP1.4: Calculate Local weights
        /*-------------------------------------------------------------------*/
        printf("STEP1.4: Calculate Local weights\n");

        // local weight for future karma for contributed light calculation (step 9, may be moved to end)
        double *d_cyberlinksLocalWeights;
        hipMalloc(&d_cyberlinksLocalWeights, linksSize*sizeof(double));
        
        get_cyberlinks_weight_by_stake<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_stakes, d_outLinksStartIndex,
            d_outLinksCount, d_outLinksUsers, d_cidsTotalOutStakes, d_cyberlinksLocalWeights
        );

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/


        // STEP2: Calculate compressed in links count
        /*-------------------------------------------------------------------*/
        printf("STEP2: Calculate compressed in links count\n");

        // commented sources was initialized before
        // uint64_t *d_inLinksStartIndex;
        // uint32_t *d_inLinksCount;
        uint64_t *d_inLinksOuts;
        uint32_t *d_compressedInLinksCount;

        // free all before rank iterations
        // hipMalloc(&d_inLinksStartIndex,      cidsSize*sizeof(uint64_t));
        // hipMalloc(&d_inLinksCount,           cidsSize*sizeof(uint32_t));
        hipMalloc(&d_inLinksOuts,           linksSize*sizeof(uint64_t));
        hipMalloc(&d_compressedInLinksCount, cidsSize*sizeof(uint32_t));   //calculated

        // hipMemcpy(d_inLinksStartIndex, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        // hipMemcpy(d_inLinksCount,      inLinksCount,      cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_inLinksOuts,       inLinksOuts,      linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        get_compressed_in_links_count<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_inLinksStartIndex, d_inLinksCount, d_inLinksOuts, d_compressedInLinksCount
        );

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/


        // STEP3: Calculate world entropy
        /*-------------------------------------------------------------------*/
        printf("STEP3: Calculate world entropy\n");

        double *d_sumswd;
        hipMalloc(&d_sumswd, cidsSize*sizeof(double));
        hipMemcpy(d_sumswd, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        sum_stake_with_damping_by_links<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_inLinksStartIndex,
            d_inLinksCount, d_inLinksOuts, d_swd, dampingFactor, d_sumswd);
        // thrust::device_ptr<double> QJ(d_sumswd);
        // for(uint64_t i = 0; i < 21; i++) {
        //     printf("[%d] = %f\n",i,(double)*(QJ+i));
        // }

        uint64_t *d_outLinksIns;
        hipMalloc(&d_outLinksIns, linksSize*sizeof(uint64_t));
        hipMemcpy(d_outLinksIns, outLinksIns, linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        sum_stake_with_damping_by_links<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_outLinksStartIndex,
            d_outLinksCount, d_outLinksIns, d_swd, 1-dampingFactor, d_sumswd);
        
        // thrust::device_ptr<double> QJ(d_sumswd);
        // for(uint64_t i = 0; i < 21; i++) {
        //     printf("[%d] = %f\n",i,(double)*(QJ+i));
        // }

        // calculate entropy by in/out links
        double *d_entropy;
        hipMalloc(&d_entropy, cidsSize*sizeof(double));
        hipMemcpy(d_entropy, entropy, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        calculate_entropy_by_links<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_inLinksStartIndex,
            d_inLinksCount, d_inLinksOuts, d_swd, d_sumswd, d_entropy);
                // thrust::device_ptr<double> ENT(d_entropy);
        // for(uint64_t i = 0; i < 21; i++) {
        //     printf("[%d] = %f\n",i,(double)*(ENT+i));
        // }

        calculate_entropy_by_links<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
        cidsSize, d_outLinksStartIndex,
        d_outLinksCount, d_outLinksIns, d_swd, d_sumswd, d_entropy);
        
        hipFree(d_swd);
        hipFree(d_sumswd);
        hipFree(d_outLinksIns);
        // thrust::device_ptr<double> ENT(d_entropy);
        // for(uint64_t i = 0; i < 21; i++) {
        //     printf("[%d] = %f\n",i,(double)*(ENT+i));
        // }


        // STEP4: Calculate compressed in links start indexes
        /*-------------------------------------------------------------------*/
        printf("STEP4: Calculate compressed in links start indexes\n");

        uint32_t *compressedInLinksCount = (uint32_t*) malloc(cidsSize*sizeof(uint32_t));
        uint64_t *compressedInLinksStartIndex = (uint64_t*) malloc(cidsSize*sizeof(uint64_t));
        hipMemcpy(compressedInLinksCount, d_compressedInLinksCount, cidsSize * sizeof(uint32_t), hipMemcpyDeviceToHost);

        // calculated on CPU
        uint64_t compressedInLinksSize = get_links_start_index(
            cidsSize, compressedInLinksCount, compressedInLinksStartIndex
        );

        uint64_t *d_compressedInLinksStartIndex;
        hipMalloc(&d_compressedInLinksStartIndex, cidsSize*sizeof(uint64_t));
        hipMemcpy(d_compressedInLinksStartIndex, compressedInLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        free(compressedInLinksStartIndex);

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/


        // STEP5: Calculate compressed in links
        /*-------------------------------------------------------------------*/
        printf("STEP5: Calculate compressed in links\n");

        // uint64_t *d_inLinksUsers;
        CompressedInLink *d_compressedInLinks; //calculated

        // hipMalloc(&d_inLinksUsers,                   linksSize*sizeof(uint64_t));
        hipMalloc(&d_compressedInLinks,  compressedInLinksSize*sizeof(CompressedInLink));
        // hipMemcpy(d_inLinksUsers, inLinksUsers,      linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        get_compressed_in_links<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize,
            d_inLinksStartIndex, d_inLinksCount, d_cidsTotalOutStakes,
            d_inLinksOuts, d_inLinksUsers, d_stakes,
            d_compressedInLinksStartIndex, d_compressedInLinksCount,
            d_compressedInLinks
        );

        hipFree(d_inLinksStartIndex);
        hipFree(d_inLinksCount);
        hipFree(d_inLinksUsers);
        hipFree(d_inLinksOuts);
        hipFree(d_stakes);
        hipFree(d_cidsTotalOutStakes);

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/


        // STEP6: Calculate dangling nodes rank, and default rank
        /*-------------------------------------------------------------------*/
        printf("STEP6: Calculate dangling nodes rank, and default rank\n");

        double defaultRank = (1.0 - dampingFactor) / cidsSize;
        uint64_t danglingNodesSize = 0;
        for(uint64_t i=0; i< cidsSize; i++){
            rank[i] = defaultRank;
            if(inLinksCount[i] == 0) {
                danglingNodesSize++;
            }
        }

        double innerProductOverSize = defaultRank * ((double) danglingNodesSize / (double)cidsSize);
        double defaultRankWithCorrection = (dampingFactor * innerProductOverSize) + defaultRank; //fma point

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/


        // STEP7: Calculate Rank
        /*-------------------------------------------------------------------*/
        printf("STEP7: Calculate Rank\n");

        double *d_rank, *d_prevRank;
        hipMalloc(&d_rank,     cidsSize*sizeof(double));
        hipMalloc(&d_prevRank, cidsSize*sizeof(double));
        hipMemcpy(d_rank,     rank, cidsSize*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_prevRank, rank, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        int steps = 0;
        double change = tolerance + 1.0;
        while(change > tolerance) {
            swap(d_rank, d_prevRank);
            steps++;
        	run_rank_iteration<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
                d_compressedInLinks,
                d_prevRank, d_rank, cidsSize,
                d_compressedInLinksStartIndex, d_compressedInLinksCount,
                defaultRankWithCorrection, dampingFactor
        	);
        	change = find_max_ranks_diff(d_prevRank, d_rank, cidsSize);
        	hipDeviceSynchronize();
        }

        hipMemcpy(rank, d_rank, cidsSize * sizeof(double), hipMemcpyDeviceToHost);
        
        hipFree(d_prevRank);
        hipFree(d_compressedInLinksStartIndex);
        hipFree(d_compressedInLinksCount);
        hipFree(d_compressedInLinks);

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/
        
        
        // STEP8: Calculate Light
        /*-------------------------------------------------------------------*/
        printf("STEP8: Calculate Light\n");

        double *d_light;
        hipMalloc(&d_light, cidsSize*sizeof(double));
        hipMemcpy(d_light, light, cidsSize*sizeof(double), hipMemcpyHostToDevice);
        
        multiply_arrays<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_rank, d_entropy, d_light
        );
        
        hipMemcpy(light, d_light, cidsSize * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(entropy, d_entropy, cidsSize * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_entropy);
        hipFree(d_rank);

        printSize(usageOffset);
        /*-------------------------------------------------------------------*/
        

        // STEP9: Calculate Karma
        /*-------------------------------------------------------------------*/
        printf("STEP9: Calculate Karma\n");

        double *d_karma; //
        hipMalloc(&d_karma, stakesSize*sizeof(double)); //
        hipMemcpy(d_karma, karma, stakesSize*sizeof(double), hipMemcpyHostToDevice); //

        // double *cyberlinksLocalWeights = (double*) malloc(linksSize*sizeof(double));
        // hipMemcpy(cyberlinksLocalWeights, d_cyberlinksLocalWeights, linksSize*sizeof(double), hipMemcpyDeviceToHost);

        // calculate_karma(
        //     cidsSize,
        //     outLinksStartIndex,
        //     outLinksCount,
        //     outLinksUsers,
        //     cyberlinksLocalWeights,
        //     light,
        //     karma
        // );
        calculate_karma<<<1,1>>>(
            cidsSize,
            d_outLinksStartIndex,
            d_outLinksCount,
            d_outLinksUsers,
            d_cyberlinksLocalWeights,
            d_light,
            d_karma
        );

        hipMemcpy(karma, d_karma, stakesSize * sizeof(double), hipMemcpyDeviceToHost);
        // free(cyberlinksLocalWeights);

        printSize(usageOffset);
        /*-----------------*/


        // STEP10: Total cleaning
        /*-------------------------------------------------------------------*/
        printf("STEP10: Total cleaning!\n");

        hipFree(d_outLinksStartIndex);
        hipFree(d_outLinksCount);
        hipFree(d_outLinksUsers);
        hipFree(d_light);
        hipFree(d_karma);
        // hipFree(d_karma);
        hipFree(d_cyberlinksLocalWeights);

        printSize(usageOffset);
        /*-----------------*/
    }
};